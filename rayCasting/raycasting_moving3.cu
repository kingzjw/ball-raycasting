
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>
//#include "device_launch_parameters.h"
//#include <helper_cuda.h>
////#include "sm_20_atomic_functions.h"
//
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <stdio.h>
//
//#define REAL float
////#define USE_CONST_MEM
//#define HANDLE_ERROR checkCudaErrors
//
//float   elapsedTime;
//#define START_GPU {\
//elapsedTime = 0.0;\
//cudaEvent_t     start, stop;\
//checkCudaErrors(cudaEventCreate(&start)); \
//checkCudaErrors(cudaEventCreate(&stop));\
//checkCudaErrors(cudaEventRecord(start, 0));\
//
//#define END_GPU \
//checkCudaErrors(cudaEventRecord(stop, 0));\
//checkCudaErrors(cudaEventSynchronize(stop));\
//checkCudaErrors(cudaEventElapsedTime(&elapsedTime, start, stop)); \
//printf("GPU Time used:  %3.2f ms\n", elapsedTime);\
//checkCudaErrors(cudaEventDestroy(start));\
//checkCudaErrors(cudaEventDestroy(stop));}
//
//#define START_CPU {\
//double start = omp_get_wtime();
//
//#define END_CPU \
//double end = omp_get_wtime();\
//double duration = end - start;\
//printf("CPU Time used: %3.1f ms\n", duration * 1000);}
//
////############################################################################
//#ifdef _WIN64
//#define GLUT_NO_LIB_PRAGMA
//#pragma comment (lib, "opengl32.lib")
//#pragma comment (lib, "glut64.lib")
//#endif //_WIN64
//
///* On Windows, include the local copy of glut.h and glext.h */
//#include "GL/glut.h"
//#include "GL/glext.h"
//#define GET_PROC_ADDRESS( str ) wglGetProcAddress( str )
//
////----------------------��װ��bitmap��------------------------------
//struct CPUAnimBitmap {
//	//��������
//	unsigned char    *pixels;
//	int     width, height;
//	//һ��ָ��
//	void    *dataBlock;
//
//	//���Զ�̬�����ú�����ָ��
//	void(*fAnim)(void*, int);
//	void(*animExit)(void*);
//	void(*clickDrag)(void*, int, int, int, int);
//	int     dragStartX, dragStartY;
//	
//	CPUAnimBitmap(int w, int h, void *d = NULL) {
//		width = w;
//		height = h;
//		//r g b alph
//		pixels = new unsigned char[width * height * 4];
//		dataBlock = d;
//		clickDrag = NULL;
//	}
//
//	~CPUAnimBitmap() {
//		delete[] pixels;
//	}
//
//	unsigned char* get_ptr(void) const { return pixels; }
//	long image_size(void) const { return width * height * 4; }
//
//	void click_drag(void(*f)(void*, int, int, int, int)) {
//		clickDrag = f;
//	}
//
//	//��Ⱦ���ͼƬ
//	//input: f����ʹ��GPU����õ�bitmap��ͼƬ�ĺ���
//	//		 e��cuda ��������
//	void anim_and_exit(void(*f)(void*, int), void(*e)(void*)) {
//		CPUAnimBitmap**   bitmap = get_bitmap_ptr();
//		*bitmap = this;
//		fAnim = f;
//		animExit = e;
//		// a bug in the Windows GLUT implementation prevents us from
//		// passing zero arguments to glutInit()
//		int c = 1;
//		char* dummy = "";
//		glutInit(&c, &dummy);
//		glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
//		glutInitWindowSize(width, height);
//		glutCreateWindow("bitmap");
//		glutKeyboardFunc(Key);
//		glutDisplayFunc(Draw);
//
//		if (clickDrag != NULL)
//			glutMouseFunc(mouse_func);
//
//		//glutIdleFunc����ȫ�ֵĻص���������û�д����¼�����ʱ��
//		//GLUT�����ܿ���ִ�к�̨�������������������
//		//������ã����idle function�ᱻ���ϵ��ã�ֱ���д����¼�������
//		glutIdleFunc(idle_func);
//		glutMainLoop();
//	}
//
//	// static method used for glut callbacks
//	static CPUAnimBitmap** get_bitmap_ptr(void) {
//		static CPUAnimBitmap*   gBitmap;
//		return &gBitmap;
//	}
//
//	// static method used for glut callbacks
//	static void mouse_func(int button, int state,
//		int mx, int my) {
//		if (button == GLUT_LEFT_BUTTON) {
//			CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//			if (state == GLUT_DOWN) {
//				bitmap->dragStartX = mx;
//				bitmap->dragStartY = my;
//			}
//			else if (state == GLUT_UP) {
//				bitmap->clickDrag(bitmap->dataBlock,
//					bitmap->dragStartX,
//					bitmap->dragStartY,
//					mx, my);
//			}
//		}
//	}
//
//	// static method used for glut callbacks
//	static void idle_func(void) {
//		static int ticks = 1;
//		CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//		bitmap->fAnim(bitmap->dataBlock, ticks++);
//		glutPostRedisplay();
//	}
//
//	// static method used for glut callbacks
//	static void Key(unsigned char key, int x, int y) {
//		switch (key) {
//		case 27:
//			CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//			bitmap->animExit(bitmap->dataBlock);
//			//delete bitmap;
//			exit(0);
//		}
//	}
//
//	// static method used for glut callbacks
//	static void Draw(void) {
//		CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//		glClearColor(0.0, 0.0, 0.0, 1.0);
//		glClear(GL_COLOR_BUFFER_BIT);
//		glDrawPixels(bitmap->width, bitmap->height, GL_RGBA, GL_UNSIGNED_BYTE, bitmap->pixels);
//		glutSwapBuffers();
//	}
//};
//
////ͼƬ������ֵ
//#define DIM 1024
//#define rnd( x ) (x * rand() / RAND_MAX)
//#define INF     2e10f
//
////----------------------------��װ��һ����-------------------------------
//struct Sphere {
//	REAL   r, b, g;
//	REAL   radius;
//	//С���λ��
//	REAL   x, y, z;
//	//ÿһ֡С����ƶ����ٶ�
//	REAL dx, dy, dz;
//	bool isCrash;
//	//������ ox,oy�������صĹ��ߣ��Ƿ����������ཻ��
//	//��������������ཻ����ô���������������������������������ľ��롣
//	//�����Ͷ�������ཻ��ֻ��¼��ӽ����������Żᱻ������
//	__device__ REAL hit(REAL ox, REAL oy, REAL *n) {
//		REAL dx = ox - x;
//		REAL dy = oy - y;
//		//����С������İ뾶��ʱ�򣬲��ܺ������ཻ
//		if (dx*dx + dy*dy < radius*radius) {
//			REAL dz = sqrtf(radius*radius - dx*dx - dy*dy);
//			*n = dz / sqrtf(radius * radius);
//			return dz + z;
//		}
//		//����Զ
//		return -INF;
//	}
//};
//
////------------С����ײ�ĸ���----------
//#define SPHERES 2000
//
//int *d_crashnum, *h_crashnum;
//
//#ifdef USE_CONST_MEM
//__constant__ Sphere d_spheres[SPHERES];
//#else
//Sphere  *d_spheres;
//#endif
//
////------------------------cuda kernel --------------------------
//
//#define STEP_SIZE REAL(20.0)
//
////�����ײ��С��ĸ���
//__global__ void crash_sharedMemory(Sphere *s, int num_sphere, int*d_crashnum)
//{
//	//--------�������ݵ��ڴ��� һ��block����---------
//	__shared__ Sphere rowSphere[32];
//	__shared__ Sphere colSphere[32];
//	int rowId1 = blockIdx.x * blockDim.x;
//	for (int i = rowId1; i < rowId1 + blockDim.x; i++)
//	{
//		if (i >= num_sphere)
//			break;
//		rowSphere[i-rowId1] = s[i];
//	}
//
//	int colId1 = blockIdx.y * blockDim.y;
//	for (int i = colId1; i < colId1 + blockDim.y; i++)
//	{
//		if (i >= num_sphere)
//			break;
//		colSphere[i- colId1] = s[i];
//	}
//	//�ȴ����ݿ��Լ���ok
//	__syncthreads();
//
//	//�õ�������ײС������
//	int s1 = threadIdx.x + blockIdx.x * blockDim.x;
//	int s2 = threadIdx.y + blockIdx.y * blockDim.y;
//
//	//�����Ϊx,y������С�������ײ���,�Գƾ��󣬼���һ��ľ���
//	if (s1 < num_sphere && s2 < num_sphere && s1 < s2)
//	{
//		REAL dx = rowSphere[threadIdx.x].x - colSphere[threadIdx.y].x;
//		REAL dy = rowSphere[threadIdx.x].y - colSphere[threadIdx.y].y;
//		REAL dz = rowSphere[threadIdx.x].z - colSphere[threadIdx.y].z;
//		REAL totalRadius = rowSphere[threadIdx.x].radius + colSphere[threadIdx.y].radius;
//		//�ж��Ƿ���ײ
//		if (dx*dx + dy*dy + dz*dz <= totalRadius * totalRadius)
//		{
//			s[s1].isCrash = true;
//			s[s2].isCrash = true;
//			atomicAdd(d_crashnum, 1);
//		}
//	}
//
//	__syncthreads();
//}
//
////�����������ڵ�λ��
//__global__ void kernelMoving(Sphere *s, int len)
//{
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	//�Ե�x �����壬���������ڵ�λ��
//	while (x < len) {
//		
//		s[x].isCrash = false;
//		s[x].x += s[x].dx;
//		s[x].y += s[x].dy;
//		s[x].z += s[x].dz;
//		x += gridDim.x*blockDim.x;
//	}
//}
//
//#ifdef USE_CONST_MEM
//__global__ void kernel(unsigned char *ptr) {
//#else
//__global__ void kernel(Sphere *d_spheres, unsigned char *ptr) {
//#endif
//	//�õ�pixel �����ص�λ�á�
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	//���ǵڼ�������
//	int offset = x + y * blockDim.x * gridDim.x;
//	REAL   ox = (x - DIM / 2);
//	REAL   oy = (y - DIM / 2);
//
//	REAL   r = 0, g = 0, b = 0;
//	REAL   maxz = -INF;
//	for (int i = 0; i < SPHERES; i++) {
//		REAL   n;
//		REAL   t = d_spheres[i].hit(ox, oy, &n);
//		if (t > maxz) {
//			REAL fscale = n;
//			if (d_spheres[i].isCrash)
//			{
//				r = 1.0f *fscale;
//				g = 0.0f*fscale;
//				b = 0.0f*fscale;
//			}
//			else
//			{
//				r = d_spheres[i].r * fscale;
//				g = d_spheres[i].g * fscale;
//				b = d_spheres[i].b * fscale;
//				maxz = t;
//			}
//		}
//	}
//
//	ptr[offset * 4 + 0] = (int)(r * 255);
//	ptr[offset * 4 + 1] = (int)(g * 255);
//	ptr[offset * 4 + 2] = (int)(b * 255);
//	ptr[offset * 4 + 3] = 255;
//}
//
//// globals needed by the update routine
//struct DataBlock {
//	//��� gpu �е�bitmap ������
//	unsigned char   *dev_bitmap;
//	//cpu�д��bitmap ������
//	CPUAnimBitmap  *bitmap;
//};
//
//void generate_frame(DataBlock *d, int ticks) {
//	
//	// initialize all integers of a device_vector to 0
//	/*int * d_crashNumList;
//	cudaMalloc(&d_crashNumList, sizeof(int)* SPHERES);
//	cudaMemset(d_crashNumList, 0, sizeof(int)* SPHERES);*/
//	
//	float totalTime = 0.0;
//	//��С�����ײ�ļ�������0
//	HANDLE_ERROR(cudaMemset(d_crashnum, 0, sizeof(int)));
//	//��С��ĸ��� copy��host �У�����ӡ����
//	HANDLE_ERROR(cudaMemcpy(h_crashnum, d_crashnum,
//		sizeof(int), cudaMemcpyDeviceToHost));
//	printf("init num of crash:  %d\n", (*h_crashnum));
//
//	START_GPU
//
//	//------------�ƶ���С��  --2000�� ----------------
//	kernelMoving << <64, 32 >> > (d_spheres, SPHERES);
//	END_GPU
//	totalTime += elapsedTime;
//	START_GPU
//	//--------------���С�����ײ------------------
//	//SPHERES ��2000 ��ʱ�� grid 64 * 64
//	//dim3    crashGrids(64, 64);
//	dim3    crashGrids(64, 64);
//	dim3    crashBlock(32, 32);
//	
//	crash_sharedMemory << <crashGrids, crashBlock >> > (d_spheres, SPHERES, d_crashnum);
//	
//	END_GPU
//	totalTime += elapsedTime;
//
//	//-----------��С������������һ�ŵ� bitmap--------
//	START_GPU
//	dim3    grids(DIM / 16, DIM / 16);
//	dim3    threads(16, 16);
//#ifdef USE_CONST_MEM
//	kernel << <grids, threads >> > (d->dev_bitmap);
//#else
//	kernel << <grids, threads >> > (d_spheres, d->dev_bitmap);
//#endif
//
//	END_GPU
//	totalTime += elapsedTime;
//
//	//-----��bitmap �����ݴ� device ������ host ��-----------
//	HANDLE_ERROR(cudaMemcpy(d->bitmap->get_ptr(), d->dev_bitmap,
//		d->bitmap->image_size(), cudaMemcpyDeviceToHost));
//
//	HANDLE_ERROR(cudaMemcpy(h_crashnum, d_crashnum,
//		sizeof(int), cudaMemcpyDeviceToHost));
//	printf("num of pair sphere crash:  %d\n", (*h_crashnum));
//	printf("total time:  %3.1f\n", totalTime);
//	printf("---------------------------------------------\n");
//
//}	
//
//// clean up memory allocated on the GPU
//void cleanup(DataBlock *d) {
//	HANDLE_ERROR(cudaFree(d->dev_bitmap));
//	//�ͷ�С����ײ�����Ŀռ�
//	HANDLE_ERROR(cudaFree(d_crashnum));
//	free(h_crashnum);
//}
//
////-------------------------main-------------------------------
//
//int main(void) {
//	//---------����ͼƬ�Ŀռ�----------
//	DataBlock   data;
//	CPUAnimBitmap bitmap(DIM, DIM, &data);
//	data.bitmap = &bitmap;
//
//	//����С����ײ�ļ������Ŀռ�
//	h_crashnum = (int *)malloc(sizeof(int));
//	*h_crashnum = 0;
//	
//	HANDLE_ERROR(cudaMalloc((void**)&d_crashnum, sizeof(int)));
//	HANDLE_ERROR(cudaMemcpy(d_crashnum, h_crashnum,
//		sizeof(int), cudaMemcpyHostToDevice));
//	
//	//---------����gpu�ռ�-------------
//	HANDLE_ERROR(cudaMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
//
//#ifdef USE_CONST_MEM
//#else
//	HANDLE_ERROR(cudaMalloc((void**)&d_spheres, sizeof(Sphere) * SPHERES));
//#endif
//
//	// allocate temp memory, initialize it, copy to constant
//	// memory on the GPU, then free our temp memory
//	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
//	for (int i = 0; i < SPHERES; i++) {
//		temp_s[i].r = rnd(1.0f);
//		temp_s[i].g = rnd(1.0f);
//		temp_s[i].b = rnd(1.0f);
//		
//		temp_s[i].x = rnd(1000.0f) - 500;
//		temp_s[i].y = rnd(1000.0f) - 500;
//		temp_s[i].z = rnd(1000.0f) - 500;
//		temp_s[i].radius = rnd(10.0f) + 5;
//
//		//��ʼ�� С���ƶ����ٶ�
//		temp_s[i].dx = STEP_SIZE * ((rand() / (float)RAND_MAX) * 2 - 1);
//		temp_s[i].dy = STEP_SIZE * ((rand() / (float)RAND_MAX) * 2 - 1);
//		temp_s[i].dz = STEP_SIZE * ((rand() / (float)RAND_MAX) * 2 - 1);
//	}
//
//#ifdef USE_CONST_MEM
//	HANDLE_ERROR(cudaMemcpyToSymbol(d_spheres, temp_s, sizeof(Sphere) * SPHERES));
//#else
//	HANDLE_ERROR(cudaMemcpy(d_spheres, temp_s, sizeof(Sphere)*SPHERES, cudaMemcpyHostToDevice));
//#endif
//
//	free(temp_s);
//
//	// display
//	bitmap.anim_and_exit((void(*)(void*, int))generate_frame, (void(*)(void*))cleanup);
//}