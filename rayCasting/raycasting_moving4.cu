
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>
//#include "device_launch_parameters.h"
//#include <helper_cuda.h>
////#include "sm_20_atomic_functions.h"
//
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <stdio.h>
//
//#define REAL float
////#define USE_CONST_MEM
//#define HANDLE_ERROR checkCudaErrors
//
//float   elapsedTime;
//#define START_GPU {\
//elapsedTime = 0.0;\
//cudaEvent_t     start, stop;\
//checkCudaErrors(cudaEventCreate(&start)); \
//checkCudaErrors(cudaEventCreate(&stop));\
//checkCudaErrors(cudaEventRecord(start, 0));\
//
//#define END_GPU \
//checkCudaErrors(cudaEventRecord(stop, 0));\
//checkCudaErrors(cudaEventSynchronize(stop));\
//checkCudaErrors(cudaEventElapsedTime(&elapsedTime, start, stop)); \
//printf("GPU Time used:  %3.2f ms\n", elapsedTime);\
//checkCudaErrors(cudaEventDestroy(start));\
//checkCudaErrors(cudaEventDestroy(stop));}
//
//#define START_CPU {\
//double start = omp_get_wtime();
//
//#define END_CPU \
//double end = omp_get_wtime();\
//double duration = end - start;\
//printf("CPU Time used: %3.1f ms\n", duration * 1000);}
//
////############################################################################
//#ifdef _WIN64
//#define GLUT_NO_LIB_PRAGMA
//#pragma comment (lib, "opengl32.lib")
//#pragma comment (lib, "glut64.lib")
//#endif //_WIN64
//
///* On Windows, include the local copy of glut.h and glext.h */
//#include "GL/glut.h"
//#include "GL/glext.h"
//#define GET_PROC_ADDRESS( str ) wglGetProcAddress( str )
//
////----------------------��װ��bitmap��------------------------------
//struct CPUAnimBitmap {
//	//��������
//	unsigned char    *pixels;
//	int     width, height;
//	//һ��ָ��
//	void    *dataBlock;
//
//	//���Զ�̬�����ú�����ָ��
//	void(*fAnim)(void*, int);
//	void(*animExit)(void*);
//	void(*clickDrag)(void*, int, int, int, int);
//	int     dragStartX, dragStartY;
//	
//	CPUAnimBitmap(int w, int h, void *d = NULL) {
//		width = w;
//		height = h;
//		//r g b alph
//		pixels = new unsigned char[width * height * 4];
//		dataBlock = d;
//		clickDrag = NULL;
//	}
//
//	~CPUAnimBitmap() {
//		delete[] pixels;
//	}
//
//	unsigned char* get_ptr(void) const { return pixels; }
//	long image_size(void) const { return width * height * 4; }
//
//	void click_drag(void(*f)(void*, int, int, int, int)) {
//		clickDrag = f;
//	}
//
//	//��Ⱦ���ͼƬ
//	//input: f����ʹ��GPU����õ�bitmap��ͼƬ�ĺ���
//	//		 e��cuda ������
//	void anim_and_exit(void(*f)(void*, int), void(*e)(void*)) {
//		CPUAnimBitmap**   bitmap = get_bitmap_ptr();
//		*bitmap = this;
//		fAnim = f;
//		animExit = e;
//		// a bug in the Windows GLUT implementation prevents us from
//		// passing zero arguments to glutInit()
//		int c = 1;
//		char* dummy = "";
//		glutInit(&c, &dummy);
//		glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
//		glutInitWindowSize(width, height);
//		glutCreateWindow("bitmap");
//		glutKeyboardFunc(Key);
//		glutDisplayFunc(Draw);
//
//		if (clickDrag != NULL)
//			glutMouseFunc(mouse_func);
//
//		//glutIdleFunc����ȫ�ֵĻص���������û�д����¼�����ʱ��
//		//GLUT�����ܿ���ִ�к�̨�������������������
//		//������ã����idle function�ᱻ���ϵ��ã�ֱ���д����¼�������
//		glutIdleFunc(idle_func);
//		glutMainLoop();
//	}
//
//	// static method used for glut callbacks
//	static CPUAnimBitmap** get_bitmap_ptr(void) {
//		static CPUAnimBitmap*   gBitmap;
//		return &gBitmap;
//	}
//
//	// static method used for glut callbacks
//	static void mouse_func(int button, int state,
//		int mx, int my) {
//		if (button == GLUT_LEFT_BUTTON) {
//			CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//			if (state == GLUT_DOWN) {
//				bitmap->dragStartX = mx;
//				bitmap->dragStartY = my;
//			}
//			else if (state == GLUT_UP) {
//				bitmap->clickDrag(bitmap->dataBlock,
//					bitmap->dragStartX,
//					bitmap->dragStartY,
//					mx, my);
//			}
//		}
//	}
//
//	// static method used for glut callbacks
//	static void idle_func(void) {
//		static int ticks = 1;
//		CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//		bitmap->fAnim(bitmap->dataBlock, ticks++);
//		glutPostRedisplay();
//	}
//
//	// static method used for glut callbacks
//	static void Key(unsigned char key, int x, int y) {
//		switch (key) {
//		case 27:
//			CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//			bitmap->animExit(bitmap->dataBlock);
//			//delete bitmap;
//			exit(0);
//		}
//	}
//
//	// static method used for glut callbacks
//	static void Draw(void) {
//		CPUAnimBitmap*   bitmap = *(get_bitmap_ptr());
//		glClearColor(0.0, 0.0, 0.0, 1.0);
//		glClear(GL_COLOR_BUFFER_BIT);
//		glDrawPixels(bitmap->width, bitmap->height, GL_RGBA, GL_UNSIGNED_BYTE, bitmap->pixels);
//		glutSwapBuffers();
//	}
//};
//
////ͼƬ������ֵ
//#define DIM 1024
//#define rnd( x ) (x * rand() / RAND_MAX)
//#define INF     2e10f
//
////----------------------------��װ��һ����-------------------------------
//struct Sphere {
//	REAL   r, b, g;
//	REAL   radius;
//	//С���λ��
//	REAL   x, y, z;
//	//ÿһ֡С����ƶ����ٶ�
//	REAL dx, dy, dz;
//	bool isCrash;
//	//������ ox,oy�������صĹ��ߣ��Ƿ����������ཻ��
//	//��������������ཻ����ô���������������������������������ľ��롣
//	//�����Ͷ�������ཻ��ֻ��¼��ӽ����������Żᱻ������
//	__device__ REAL hit(REAL ox, REAL oy, REAL *n) {
//		REAL dx = ox - x;
//		REAL dy = oy - y;
//		//����С������İ뾶��ʱ�򣬲��ܺ������ཻ
//		if (dx*dx + dy*dy < radius*radius) {
//			REAL dz = sqrtf(radius*radius - dx*dx - dy*dy);
//			*n = dz / sqrtf(radius * radius);
//			return dz + z;
//		}
//		//����Զ
//		return -INF;
//	}
//};
//
////------------С����ײ�ĸ���----------
//#define SPHERES 2000
//
//int *d_crashnum, *h_crashnum;
//
//#ifdef USE_CONST_MEM
//__constant__ Sphere d_spheres[SPHERES];
//#else
//Sphere  *d_spheres;
//#endif
//
////------------------------cuda kernel --------------------------
//
//#define STEP_SIZE REAL(20.0)
//
////�����ײ��С��ĸ���
//__global__ void crash(Sphere *s, int num_sphere, int*d_crashnum , int streamId , int streamNum)
//{
//	//�õ�������ײС������
//	int s1 = threadIdx.x + blockIdx.x * blockDim.x;
//	int s2 = threadIdx.y + blockIdx.y * blockDim.y;
//
//	s2 = s2 + 64 / 4 * streamId * 32;
//	//�����Ϊx,y������С�������ײ���,�Գƾ��󣬼���һ��ľ���
//	if (s2 < num_sphere && s1 < num_sphere && s2 < s1)
//	//if (s2 < num_sphere && s1 < num_sphere)
//	{
//		REAL dx = s[s1].x - s[s2].x;
//		REAL dy = s[s1].y - s[s2].y;
//		REAL dz = s[s1].z - s[s2].z;
//		REAL totalRadius = s[s1].radius + s[s2].radius;
//		//�ж��Ƿ���ײ
//		if (dx*dx + dy*dy + dz*dz <= totalRadius * totalRadius)
//		{
//			s[s1].isCrash = true;
//			s[s2].isCrash = true;
//
//			//printf("y: %d  x: %d\n", s2,s1);
//
//			atomicAdd(d_crashnum, 1);
//		}
//	}
//}
//
//__global__ void addKernel(int * num0 , int * num1, int * num2, int * num3,int * res)
//{
//	*res = *num0 + *num1 + *num2 + *num3;
//}
//
////�����������ڵ�λ��
//__global__ void kernelMoving(Sphere *s, int len)
//{
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	//�Ե�x �����壬���������ڵ�λ��
//	while (x < len) {
//		
//		s[x].isCrash = false;
//		s[x].x += s[x].dx;
//		s[x].y += s[x].dy;
//		s[x].z += s[x].dz;
//		x += gridDim.x*blockDim.x;
//	}
//}
//
//#ifdef USE_CONST_MEM
//__global__ void kernel(unsigned char *ptr) {
//#else
//__global__ void kernel(Sphere *d_spheres, unsigned char *ptr) {
//#endif
//	//�õ�pixel �����ص�λ�á�
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	//���ǵڼ�������
//	int offset = x + y * blockDim.x * gridDim.x;
//	REAL   ox = (x - DIM / 2);
//	REAL   oy = (y - DIM / 2);
//
//	REAL   r = 0, g = 0, b = 0;
//	REAL   maxz = -INF;
//	for (int i = 0; i < SPHERES; i++) {
//		REAL   n;
//		REAL   t = d_spheres[i].hit(ox, oy, &n);
//		if (t > maxz) {
//			REAL fscale = n;
//			if (d_spheres[i].isCrash)
//			{
//				r = 1.0f *fscale;
//				g = 0.0f*fscale;
//				b = 0.0f*fscale;
//			}
//			else
//			{
//				r = d_spheres[i].r * fscale;
//				g = d_spheres[i].g * fscale;
//				b = d_spheres[i].b * fscale;
//				maxz = t;
//			}
//		}
//	}
//
//	ptr[offset * 4 + 0] = (int)(r * 255);
//	ptr[offset * 4 + 1] = (int)(g * 255);
//	ptr[offset * 4 + 2] = (int)(b * 255);
//	ptr[offset * 4 + 3] = 255;
//}
//
//
//// globals needed by the update routine
//struct DataBlock {
//	//��� gpu �е�bitmap ������
//	unsigned char   *dev_bitmap;
//	//cpu�д��bitmap ������
//	CPUAnimBitmap   *bitmap;
//};
//
//
//#define streamNum 4
//cudaStream_t  stream0, stream1, stream2, stream3;
//int *crashNum0, *crashNum1, *crashNum2, *crashNum3;
//Sphere *sphere0, *sphere1, *sphere2, *sphere3;
//
//void generate_frame(DataBlock *d, int ticks) {
//	float totalTime = 0.0;
//	//��С�����ײ�ļ�������0
//	HANDLE_ERROR(cudaMemset(d_crashnum, 0, sizeof(int)));
//	//��С��ĸ��� copy��host �У�����ӡ����
//
//	START_GPU
//
//	//------------�ƶ���С��  --2000�� ----------------
//	kernelMoving << <64, 32 >> > (d_spheres, SPHERES);
//	END_GPU
//	totalTime += elapsedTime;
//
//	//----------------------------����stream handle-------------------------
//	//���ĸ���
//	START_GPU
//
//	dim3    crashGrids(64, 64 / streamNum);
//	dim3    crashBlock(32, 32);
//
//	HANDLE_ERROR(cudaMemset(crashNum0, 0, sizeof(int)));
//	HANDLE_ERROR(cudaMemset(crashNum1, 0, sizeof(int)));
//	HANDLE_ERROR(cudaMemset(crashNum2, 0, sizeof(int)));
//	HANDLE_ERROR(cudaMemset(crashNum3, 0, sizeof(int)));
//
//	cudaMemcpyAsync(crashNum0, d_crashnum, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream0);
//	cudaMemcpyAsync(crashNum1, d_crashnum, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream1);
//	cudaMemcpyAsync(crashNum2, d_crashnum, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream2);
//	cudaMemcpyAsync(crashNum3, d_crashnum, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream3);
//
//	cudaMemcpyAsync(sphere0, d_spheres, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream0);
//	cudaMemcpyAsync(sphere1, d_spheres, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream1);
//	cudaMemcpyAsync(sphere2, d_spheres, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream2);
//	cudaMemcpyAsync(sphere3, d_spheres, sizeof(Sphere) * SPHERES, cudaMemcpyDeviceToDevice, stream3);
//
//	crash << <crashGrids, crashBlock, 0, stream0 >> > (sphere0, SPHERES, crashNum0, 3, streamNum);
//	crash << <crashGrids, crashBlock, 0, stream1 >> > (sphere1, SPHERES, crashNum1, 2, streamNum);
//	crash << <crashGrids, crashBlock, 0, stream2 >> > (sphere2, SPHERES, crashNum2, 1, streamNum);
//	crash << <crashGrids, crashBlock, 0, stream3 >> > (sphere3, SPHERES, crashNum3, 0, streamNum);
//
//	//----------------------ͬ����------------------------------
//	cudaStreamSynchronize(stream0);
//	cudaStreamSynchronize(stream1);
//	cudaStreamSynchronize(stream2);
//	cudaStreamSynchronize(stream3);
//
//	/*thrust::host_vector<int> crashNumList(4);
//	crashNumList[0] = *crashNum0;
//	crashNumList[1] = *crashNum1;
//	crashNumList[2] = *crashNum2;
//	crashNumList[3] = *crashNum3;
//	int sum = thrust::reduce(crashNumList.begin(), crashNumList.end(), (int)0, thrust::plus<int>());*/
//	//printf("num of pair sphere crash:  %d\n", sum);
//
//	addKernel << <1, 1 >> > (crashNum0, crashNum1, crashNum2, crashNum3, d_crashnum);
//	//*d_crashnum = * + *crashNum1 + *crashNum2 + *crashNum3;
//	END_GPU
//
//	totalTime += elapsedTime;
//
//	//-----------��С������������һ�ŵ� bitmap--------
//	START_GPU
//	dim3    grids(DIM / 16, DIM / 16);
//	dim3    threads(16, 16);
//#ifdef USE_CONST_MEM
//	kernel << <grids, threads >> > (d->dev_bitmap);
//#else
//	kernel << <grids, threads >> > (d_spheres, d->dev_bitmap);
//#endif
//
//	END_GPU
//	totalTime += elapsedTime;
//
//	//-----��bitmap �����ݴ� device ������ host ��-----------
//	HANDLE_ERROR(cudaMemcpy(d->bitmap->get_ptr(), d->dev_bitmap,
//		d->bitmap->image_size(), cudaMemcpyDeviceToHost));
//
//	HANDLE_ERROR(cudaMemcpy(h_crashnum, d_crashnum,sizeof(int), cudaMemcpyDeviceToHost));
//	printf("num of pair sphere crash:  %d\n", (*h_crashnum));
//	printf("total time:  %3.1f\n", totalTime);
//	printf("---------------------------------------------\n");
//}	
//
//// clean up memory allocated on the GPU
//void cleanup(DataBlock *d) {
//	HANDLE_ERROR(cudaFree(d->dev_bitmap));
//	//�ͷ�С����ײ�����Ŀռ�
//	HANDLE_ERROR(cudaFree(d_crashnum));
//	free(h_crashnum);
//
//	//----------free stream-----------
//	cudaStreamDestroy(stream0);
//	cudaStreamDestroy(stream1);
//	cudaStreamDestroy(stream2);
//	cudaStreamDestroy(stream3);
//
//}
//
////-------------------------main-------------------------------
//
//int main(void) {
//	//-----------------�����Ƿ������������----------------------
//	cudaDeviceProp  prop;
//	int whichDevice;
//	cudaGetDevice(&whichDevice);
//	cudaGetDeviceProperties(&prop, whichDevice);
//	if (!prop.deviceOverlap) {
//		printf("Device will not handle overlaps, so no speed up from streams\n");
//		return;
//	}
//	else
//	{
//		printf("Device will  handle overlaps, so we can speed up from streams\n");
//	}
//
//	//----------create stream-----------
//	cudaStreamCreate(&stream0);
//	cudaStreamCreate(&stream1);
//	cudaStreamCreate(&stream2);
//	cudaStreamCreate(&stream3);
//
//	//--------------���С�����ײ------------------
//	HANDLE_ERROR(cudaMalloc(&crashNum0, sizeof(int)));
//	HANDLE_ERROR(cudaMalloc(&crashNum1, sizeof(int)));
//	HANDLE_ERROR(cudaMalloc(&crashNum2, sizeof(int)));
//	HANDLE_ERROR(cudaMalloc(&crashNum3, sizeof(int)));
//	HANDLE_ERROR(cudaMalloc(&sphere0, sizeof(Sphere) * SPHERES));
//	HANDLE_ERROR(cudaMalloc(&sphere1, sizeof(Sphere) * SPHERES));
//	HANDLE_ERROR(cudaMalloc(&sphere2, sizeof(Sphere) * SPHERES));
//	HANDLE_ERROR(cudaMalloc(&sphere3, sizeof(Sphere) * SPHERES));
//
//
//	//---------����ͼƬ�Ŀռ�----------
//	DataBlock   data;
//	CPUAnimBitmap bitmap(DIM, DIM, &data);
//	data.bitmap = &bitmap;
//
//	//����С����ײ�ļ������Ŀռ�
//	h_crashnum = (int *)malloc(sizeof(int));
//	*h_crashnum = 0;
//	
//	HANDLE_ERROR(cudaMalloc((void**)&d_crashnum, sizeof(int)));
//	HANDLE_ERROR(cudaMemcpy(d_crashnum, h_crashnum,sizeof(int), cudaMemcpyHostToDevice));
//	//---------����gpu�ռ�-------------
//	HANDLE_ERROR(cudaMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
//
//#ifdef USE_CONST_MEM
//#else
//	HANDLE_ERROR(cudaMalloc((void**)&d_spheres, sizeof(Sphere) * SPHERES));
//#endif
//
//	// allocate temp memory, initialize it, copy to constant
//	// memory on the GPU, then free our temp memory
//	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
//	for (int i = 0; i < SPHERES; i++) {
//		temp_s[i].r = rnd(1.0f);
//		temp_s[i].g = rnd(1.0f);
//		temp_s[i].b = rnd(1.0f);
//		
//		temp_s[i].x = rnd(1000.0f) - 500;
//		temp_s[i].y = rnd(1000.0f) - 500;
//		temp_s[i].z = rnd(1000.0f) - 500;
//		temp_s[i].radius = rnd(10.0f) + 5;
//
//		//��ʼ�� С���ƶ����ٶ�
//		temp_s[i].dx = STEP_SIZE * ((rand() / (float)RAND_MAX) * 2 - 1);
//		temp_s[i].dy = STEP_SIZE * ((rand() / (float)RAND_MAX) * 2 - 1);
//		temp_s[i].dz = STEP_SIZE * ((rand() / (float)RAND_MAX) * 2 - 1);
//	}
//
//#ifdef USE_CONST_MEM
//	HANDLE_ERROR(cudaMemcpyToSymbol(d_spheres, temp_s, sizeof(Sphere) * SPHERES));
//#else
//	HANDLE_ERROR(cudaMemcpy(d_spheres, temp_s, sizeof(Sphere)*SPHERES, cudaMemcpyHostToDevice));
//#endif
//
//	free(temp_s);
//
//	// display
//	bitmap.anim_and_exit((void(*)(void*, int))generate_frame, (void(*)(void*))cleanup);
//}